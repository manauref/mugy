#include "hip/hip_runtime.h"
/* mugy: initialization_dev.c
 *
 * Functions used to initialize the device (GPU).
 *
 */

extern "C" {
#include "mh_initialization_dev.h"
#include "mh_utilities_dev.h"
}

void device_init_dev(struct mugy_comms *comms) {

  // Set the device of this MPI rank.
  int devCount = 0;
  checkCudaErrors(hipGetDeviceCount(&devCount));
  mint mpiRank = comms->world.rank;
  mint devID = mpiRank % devCount;
  checkCudaErrors(hipSetDevice(devID));
  printf("  My rank: %d | # of GPUs: %d | my GPU: %d\n", mpiRank, devCount, devID);
  printf("\n");

  // Get device properties.
  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

  printf("  Device properties:\n");
  printf("    global memory = %zu\n",deviceProp.totalGlobalMem);
  printf("    shared memory/block = %zu\n",deviceProp.sharedMemPerBlock);
  printf("    warp size = %d\n",deviceProp.warpSize);
  printf("    max threads/block = %d\n",deviceProp.maxThreadsPerBlock);
  printf("    major compute capability = %d\n",deviceProp.major);
  printf("    minor compute capability = %d\n",deviceProp.minor);
  printf("\n");

}

