/* mugy: alloc_dev.cu

   Functions used to allocate arrays on device (GPU).
*/

extern "C" {
#include "mh_utilities_dev.h"
#include "mh_data_dev.h"
#include "mh_alloc_dev.h"
}

real* alloc_realArray_dev(int numElements) {
  real *out_p;
  checkCudaErrors(hipMalloc(&out_p, numElements*sizeof(real)));
  return out_p;
}
// MF 2023/03/29: Returning a void* because C doesn't know cuCumplex/cufourier the type.
void* alloc_fourierArray_dev(const mint numElements) {
  cufourier *out_p;
  checkCudaErrors(hipMalloc(&out_p, numElements*sizeof(cufourier)));
  return out_p;
}

void free_realMoments_dev(real *mom_dev) {
  checkCudaErrors(hipFree(mom_dev));  // Free free memory.
}
void free_fourierMoments_dev(void *momk_dev) {
  checkCudaErrors(hipFree((cufourier*) momk_dev));  // Free device memory.
}
