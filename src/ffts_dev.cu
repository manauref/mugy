/* mugy: ffts_dev.cu
 *
 * Device FFT methods.
 *
 */

extern "C" {
#include "mh_ffts.h"
#include "mh_ffts_dev.h"
#include "mh_utilities.h"
}
#include "mh_ffts_dev_priv.h"

struct mugy_fft_fam_dev *mugy_fft_init_dev(struct mugy_grid *grid, struct mugy_pop popL, struct mugy_comms *comms) {

  struct mugy_fft_fam_dev *ffts = (struct mugy_fft_fam_dev *) malloc(sizeof(struct mugy_fft_fam_dev));

  struct mugy_fft_dev *cfft;  // Temp pointer for convenience.

  // ....... Setup for 2D FFTs of a single field ......... //
  ffts->xy = (struct mugy_fft_dev *) malloc(sizeof(struct mugy_fft_dev));
  cfft = ffts->xy;

  CUDA_RT_CALL(hipStreamCreateWithFlags(&cfft->stream, hipStreamNonBlocking));

  size_t workSizes_r2c[1], workSizes_c2r[1];
  const mint fftDim = 2;
  long long int fftSize[fftDim], fftNum;
  long long int inembed[fftDim], istride, idist;
  long long int onembed[fftDim], ostride, odist;
  fftSize[0] = grid->global.deal.dual.Nx[0];
  fftSize[1] = grid->global.deal.dual.Nx[1];
  fftNum     = grid->local.deal.Nekx[2];
  istride    = 1;
  ostride    = 1;

  inembed[0] = grid->local.deal.dual.Nx[0];
  inembed[1] = grid->local.deal.dual.Nx[1];
  idist      = grid->local.deal.dual.NxyTot;
  onembed[0] = grid->local.deal.Nekx[0];
  onembed[1] = grid->local.deal.Nekx[1];
  odist      = grid->local.deal.NekxyTot;
  CUFFT_CALL(hipfftCreate(&cfft->plan_r2c));
  CUFFT_CALL(hipfftXtMakePlanMany(cfft->plan_r2c, fftDim, fftSize,
    inembed, istride, idist, MUGY_CUFFT_REAL,
    onembed, ostride, odist, MUGY_CUFFT_FOURIER,
    fftNum, workSizes_r2c, MUGY_CUFFT_EXEC_FOURIER));
  CUFFT_CALL(hipfftSetStream(cfft->plan_r2c, cfft->stream));

  inembed[0] = grid->local.deal.Nekx[0];
  inembed[1] = grid->local.deal.Nekx[1];
  idist      = grid->local.deal.NekxyTot;
  onembed[0] = grid->local.deal.dual.Nx[0];
  onembed[1] = grid->local.deal.dual.Nx[1];
  odist      = grid->local.deal.dual.NxyTot;
  CUFFT_CALL(hipfftCreate(&cfft->plan_c2r));
  CUFFT_CALL(hipfftXtMakePlanMany(cfft->plan_c2r, fftDim, fftSize,
    inembed, istride, idist, MUGY_CUFFT_FOURIER,
    onembed, ostride, odist, MUGY_CUFFT_REAL,
    fftNum, workSizes_c2r, MUGY_CUFFT_EXEC_FOURIER));
  CUFFT_CALL(hipfftSetStream(cfft->plan_c2r, cfft->stream));

  // Allocate buffers.
  // Switch these mallocs to mugy functions.
  CUDA_RT_CALL(hipMalloc(&cfft->rbuf, grid->local.deal.dual.NxTot*sizeof(real)));
  CUDA_RT_CALL(hipMalloc(&cfft->kbuf, grid->local.deal.NekxTot*sizeof(mugy_cufft_fourier_t)));

  cfft->normFac = 1./((real)grid->global.deal.dual.NxyTot);
  cfft->forwardNorm = false;  // This FFT is only used for ICs given in real-space.

  // ....... End setup for 2D FFTs of a single field ......... //

  // ....... Setup for xy FFTs of all moments ......... //
  ffts->mom_xy = (struct mugy_fft_dev *) malloc(sizeof(struct mugy_fft_dev));
  cfft = ffts->mom_xy;

  CUDA_RT_CALL(hipStreamCreateWithFlags(&cfft->stream, hipStreamNonBlocking));

  fftNum     = popL.numMomentsTot*grid->local.deal.Nekx[2];

  inembed[0] = grid->local.deal.dual.Nx[0];
  inembed[1] = grid->local.deal.dual.Nx[1];
  idist      = grid->local.deal.dual.NxyTot;
  onembed[0] = grid->local.deal.Nekx[0];
  onembed[1] = grid->local.deal.Nekx[1];
  odist      = grid->local.deal.NekxyTot;
  CUFFT_CALL(hipfftCreate(&cfft->plan_r2c));
  CUFFT_CALL(hipfftXtMakePlanMany(cfft->plan_r2c, fftDim, fftSize,
    inembed, istride, idist, MUGY_CUFFT_REAL,
    onembed, ostride, odist, MUGY_CUFFT_FOURIER,
    fftNum, workSizes_r2c, MUGY_CUFFT_EXEC_FOURIER));
  CUFFT_CALL(hipfftSetStream(cfft->plan_r2c, cfft->stream));

  inembed[0] = grid->local.deal.Nekx[0];
  inembed[1] = grid->local.deal.Nekx[1];
  idist      = grid->local.deal.NekxyTot;
  onembed[0] = grid->local.deal.dual.Nx[0];
  onembed[1] = grid->local.deal.dual.Nx[1];
  odist      = grid->local.deal.dual.NxyTot;
  CUFFT_CALL(hipfftCreate(&cfft->plan_c2r));
  CUFFT_CALL(hipfftXtMakePlanMany(cfft->plan_c2r, fftDim, fftSize,
    inembed, istride, idist, MUGY_CUFFT_FOURIER,
    onembed, ostride, odist, MUGY_CUFFT_REAL,
    fftNum, workSizes_c2r, MUGY_CUFFT_EXEC_FOURIER));
  CUFFT_CALL(hipfftSetStream(cfft->plan_c2r, cfft->stream));

  // Allocate buffers.
  // Switch these mallocs to mugy functions.
  CUDA_RT_CALL(hipMalloc(&cfft->rbuf, popL.numMomentsTot*grid->local.deal.dual.NxTot*sizeof(real)));
  CUDA_RT_CALL(hipMalloc(&cfft->kbuf, popL.numMomentsTot*grid->local.deal.NekxTot*sizeof(mugy_cufft_fourier_t)));

  cfft->normFac = 1./((real)grid->global.deal.dual.NxyTot);
  cfft->forwardNorm = false;  // This FFT is only used for ICs given in real-space.

  // ....... End setup for xy FFTs of all moments ......... //

  return ffts;
}

void mugy_fft_c2r_dev(struct mugy_fft_fam_dev *ffts, struct mugy_array *fOut, struct mugy_array *fkIn, enum mugy_fft_type ttype) {

  struct mugy_fft_dev *cfft;  // Temp pointer for convenience.
  if (ttype == mugy_fft_xy)
    cfft = ffts->xy;
  else if (ttype == mugy_fft_mom_xy)
    cfft = ffts->mom_xy;
  else
    abortSimulation(" mugy_fft_c2r_dev: fft type not supported! Terminating...\n");

  // Copy data into buffer.
  mugy_memcpy(cfft->kbuf, fkIn->dev, fkIn->nelemsz, MUGY_DEVICE2DEVICE);

  // Inverse FFT.
  CUFFT_CALL(hipfftXtExec(cfft->plan_c2r, cfft->kbuf, cfft->rbuf, HIPFFT_BACKWARD));

  CUDA_RT_CALL(hipStreamSynchronize(cfft->stream));

  // Copy data from buffer.
  mugy_memcpy(fOut->dev, cfft->rbuf, fOut->nelemsz, MUGY_DEVICE2DEVICE);

  // Apply the nonunitary normalization.
  if (!cfft->forwardNorm)
    mugy_array_scale(fOut, cfft->normFac, MUGY_DEVICE_CALC);
  
}

void mugy_fft_r2c_dev(struct mugy_fft_fam_dev *ffts, struct mugy_array *fkOut, struct mugy_array *fIn, enum mugy_fft_type ttype) {

  struct mugy_fft_dev *cfft;  // Temp pointer for convenience.
  if (ttype == mugy_fft_xy)
    cfft = ffts->xy;
  else if (ttype == mugy_fft_mom_xy)
    cfft = ffts->mom_xy;
  else
    abortSimulation(" mugy_fft_r2c_dev: fft type not supported! Terminating...\n");

  // Copy data into buffer.
  mugy_memcpy(cfft->rbuf, fIn->dev, fIn->nelemsz, MUGY_DEVICE2DEVICE);

  // Forward FFT.
  CUFFT_CALL(hipfftXtExec(cfft->plan_r2c, cfft->rbuf, cfft->kbuf, HIPFFT_FORWARD));

  CUDA_RT_CALL(hipStreamSynchronize(cfft->stream));

  // Copy data from buffer.
  mugy_memcpy(fkOut->dev, cfft->kbuf, fkOut->nelemsz, MUGY_DEVICE2DEVICE);

  // Apply the nonunitary normalization.
  if (cfft->forwardNorm)
    mugy_array_scale(fkOut, cfft->normFac, MUGY_DEVICE_CALC);
  
}

void mugy_fft_terminate_dev(struct mugy_fft_fam_dev *ffts) {

  struct mugy_fft_dev *cfft = ffts->xy;  // Temp pointer for convenience.

  cfft = ffts->xy;
  CUDA_RT_CALL(hipStreamDestroy(cfft->stream));
  CUFFT_CALL(hipfftDestroy(cfft->plan_r2c));
  CUFFT_CALL(hipfftDestroy(cfft->plan_c2r));
  CUDA_RT_CALL(hipFree(cfft->rbuf));
  CUDA_RT_CALL(hipFree(cfft->kbuf));
  free(cfft);

  cfft = ffts->mom_xy;
  CUDA_RT_CALL(hipStreamDestroy(cfft->stream));
  CUFFT_CALL(hipfftDestroy(cfft->plan_r2c));
  CUFFT_CALL(hipfftDestroy(cfft->plan_c2r));
  CUDA_RT_CALL(hipFree(cfft->rbuf));
  CUDA_RT_CALL(hipFree(cfft->kbuf));
  free(cfft);

  free(ffts);
}
